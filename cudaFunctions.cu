#include "hip/hip_runtime.h"
/*
 * cudaFunctions.cu
 *
 *  Created on: Jun 4, 2012
 *      Author: petr
 */

#include <hip/hip_runtime.h>
#include "cudaFunctions.h"

__global__ void makeSumOfHashOnDevice(unsigned* hashPart2Sha1) {
    int i = threadIdx.x;

    if( i > 0 && i < HASH_SIZE ) {
        hashPart2Sha1[0] += hashPart2Sha1[i];
    }

}

void cudaMakeSumOfHash(unsigned* hashPart2Sha1) {

    unsigned* d_A;
    int size = sizeof(unsigned) * HASH_SIZE;

    hipMalloc(&d_A, size);
    hipMemcpy(d_A, hashPart2Sha1, size, hipMemcpyHostToDevice);

    makeSumOfHashOnDevice<<<1, HASH_SIZE>>>(d_A);

    hipMemcpy(hashPart2Sha1, d_A, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
}
